#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "snn_library.h"
#include "neuron_models/GPU_lif_neuron.cuh"

#define THR_PER_BLOCK 1024 

/**
D = A * B + C kalkulua egiten duen kernela
*/
__global__ void cuda_add_dot_matrix(int rowsAC, int colsBC, int colsArowsB, float *A, float *B, float *C, float *D)
{
    //lortu hariaren identifikadorea
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;

    //hariak kalkulatu behar duen Dko elementuaren errenkada eta zutabea
    int zutab = i%colsBC;
    int errenk = i/colsBC;

    //egiaztatu hariak kalkulua egin behar duela
    if(i<(rowsAC * colsBC))
    {
        //D kalkulatzeko Ako errenkada eta Bko zutabea prozesatu
        for(j=0; j<colsArowsB; j++)  
            D[i]+=A[errenk * colsArowsB + j]*B[j * colsBC + zutab];

        D[i] += C[i];
    }
}


__global__ void cuda_simulation_step_lif_neuron(int rowsAC, int colsBC, int colsArowsB, float *A, float *B, float *C, float *D)
{
    //lortu hariaren identifikadorea
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;

    //hariak kalkulatu behar duen Dko elementuaren errenkada eta zutabea
    int zutab = i%colsBC;
    int errenk = i/colsBC;

    //egiaztatu hariak kalkulua egin behar duela
    if(i<(rowsAC * colsBC))
    {
        //D kalkulatzeko Ako errenkada eta Bko zutabea prozesatu
        for(j=0; j<colsArowsB; j++)  
            D[i]+=A[errenk * colsArowsB + j]*B[j * colsBC + zutab];

        D[i] += C[i];
    }
}


/**
GPUko memoriara mugitu matrizeak eta jaurti kernela
*/
double process_simulation_lif_neuron(spiking_nn_t *snn, int n, int m, int time_steps){
    printf("Running CUDA code, YUJUUUU\n");
    
    hipEvent_t start, stop;
    
    // list of neurons and synapses
    lif_neuron_t *d_lif_neurons; 
    synapse_t *d_synapses;

    // cuda things
    float milliseconds = 0;
    int thr_per_blk_neurons, blk_in_grid_neurons, thr_per_blk_synapses, blk_in_grid_synapses;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // reserve memory for neurons and synapses lists
    //gpuErrchk(hipMalloc(&d_lif_neurons, snn->n_neurons * sizeof(lif_neuron_t)));
    //gpuErrchk(hipMalloc(&d_synapses, snn->n_synapses * sizeof(synapse_t)));
    hipMalloc(&d_lif_neurons, snn->n_neurons * sizeof(lif_neuron_t));
    hipMalloc(&d_synapses, snn->n_synapses * sizeof(synapse_t));

    // reserve memory for each neuron synapse list indexes
    for(int i = 0; i<snn->n_neurons; i++){
        //gpuErrchk(hipMalloc(&d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int)));
        hipMalloc(&d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int));
        //gpuErrchk(hipMalloc(&d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int)));
        hipMalloc(&d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int));
    }

    // reserve memory for synapse pointers
    for(int i = 0; i<snn->n_synapses; i++){
        //gpuErrchk(hipMalloc(&d_synapses[i].l_spike_times, snn->synapses[i].max_spikes * sizeof(int)));
        hipMalloc(&d_synapses[i].l_spike_times, snn->synapses[i].max_spikes * sizeof(int));
        //gpuErrchk(hipMalloc(&d_synapses[i].pre_synaptic_lif_neuron, sizeof(lif_neuron_t)));
        hipMalloc(&d_synapses[i].pre_synaptic_lif_neuron, sizeof(lif_neuron_t));
        //gpuErrchk(hipMalloc(&d_synapses[i].post_synaptic_lif_neuron, sizeof(lif_neuron_t)));
        hipMalloc(&d_synapses[i].post_synaptic_lif_neuron, sizeof(lif_neuron_t));
        //d_synapse[i].learning_rule = snn->synapses[i].learning_rule;
    }


    // copy information to gpu
    //gpuErrchk(hipMemcpy(d_lif_neurons, snn->lif_neurons, snn->n_neurons * sizeof(lif_neuron_t), hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(d_synapses, snn->synapses, snn->n_synapses * sizeof(synapse_t), hipMemcpyHostToDevice));
    hipMemcpy(d_lif_neurons, snn->lif_neurons, snn->n_neurons * sizeof(lif_neuron_t), hipMemcpyHostToDevice);
    hipMemcpy(d_synapses, snn->synapses, snn->n_synapses * sizeof(synapse_t), hipMemcpyHostToDevice);

    // copy info of neurons (ONLY POINTERS; HOW IS THE REST OF INFORMATION PASSED?)
    for(int i = 0; i<snn->n_neurons; i++){
        hipMemcpy(d_lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].input_synapse_indexes, snn->lif_neurons[i].n_input_synapse * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].output_synapse_indexes, snn->lif_neurons[i].n_output_synapse * sizeof(int), hipMemcpyHostToDevice);
   }

    // reserve memory for synapse pointers
    for(int i = 0; i<snn->n_synapses; i++){
        hipMemcpy(&d_synapses[i].l_spike_times, snn->synapses[i].l_spike_times, snn->synapses[i].max_spikes * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(&d_synapses[i].pre_synaptic_lif_neuron, snn->synapses[i].pre_synaptic_lif_neuron, sizeof(lif_neuron_t), hipMemcpyHostToDevice);
        hipMemcpy(&d_synapses[i].post_synaptic_lif_neuron, snn->synapses[i].post_synaptic_lif_neuron, sizeof(lif_neuron_t), hipMemcpyHostToDevice);
    }

    return 0.0;

    // grid for neurons kernel launching
    //thr_per_blk_neurons = colsBC; //hari bakoitzean emaitzeko matrizearen osagai bat kalkulatzen da (errenkada x zutabea)
    //blk_in_grid_neurons = rowsAC; //bloke bakoitzean A-ren errenkada bat

    // grid for synapses kernel launching
    //thr_per_blk_synapses = colsBC; //hari bakoitzean emaitzeko matrizearen osagai bat kalkulatzen da (errenkada x zutabea)
    //blk_in_grid_synapses = rowsAC;

    //blokeko gehienez 1024 hari
    /*if(colsBC > 1024)
    {
        int total = rowsAC * colsBC;
        blk_in_grid = total / 1024;
        thr_per_blk = 1024;
    }


    // simulation loop
    //for()
        // launch neuron input synapse kernel
        // launch neuron output synapse kernel
        // launch synapse learning

        // store information?
        
    // launch kernel
    gpuErrchk(hipEventRecord(start));
    cuda_add_dot_matrix<<<blk_in_grid, thr_per_blk>>>(rowsAC, colsBC, colsArowsB, d_A, d_B, d_C, d_D);
    gpuErrchk(hipEventRecord(stop));

   //Kopiatu D GPUko memoriatik CPUra
    hipMemcpy(D, d_D, rowsAC * colsBC * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    //Askatu GPUko memoria
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return(milliseconds);*/
}


/*double add_dot_matrix_GPU(int rowsAC, int colsBC, int colsArowsB, float* A, float* B, float* C, float* D)
{   
    hipEvent_t start, stop;
    float *d_A, *d_B, *d_C, *d_D;
    float milliseconds = 0;
    int thr_per_blk, blk_in_grid;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //erreserbatu memorian GPUan
    gpuErrchk(hipMalloc(&d_A, rowsAC * colsArowsB * sizeof(float)));
    gpuErrchk(hipMalloc(&d_B, colsArowsB * colsBC * sizeof(float)));
    gpuErrchk(hipMalloc(&d_C, rowsAC * colsBC * sizeof(float)));
    gpuErrchk(hipMalloc(&d_D, rowsAC * colsBC * sizeof(float)));

    //kopiatu A, B eta C matrizeak GPUko memorian
    gpuErrchk(hipMemcpy(d_A, A, rowsAC * colsArowsB * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B, B, colsArowsB * colsBC * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C, C, rowsAC * colsBC * sizeof(float), hipMemcpyHostToDevice));

    //Sareta eta blokeen egitura zehaztu
    thr_per_blk = colsBC; //hari bakoitzean emaitzeko matrizearen osagai bat kalkulatzen da (errenkada x zutabea)
    blk_in_grid = rowsAC; //bloke bakoitzean A-ren errenkada bat

    //blokeko gehienez 1024 hari
    if(colsBC > 1024)
    {
        int total = rowsAC * colsBC;
        blk_in_grid = total / 1024;
        thr_per_blk = 1024;
    }

    //jaurti kernela
    gpuErrchk(hipEventRecord(start));
    cuda_add_dot_matrix<<<blk_in_grid, thr_per_blk>>>(rowsAC, colsBC, colsArowsB, d_A, d_B, d_C, d_D);
    gpuErrchk(hipEventRecord(stop));

   //Kopiatu D GPUko memoriatik CPUra
    hipMemcpy(D, d_D, rowsAC * colsBC * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    //Askatu GPUko memoria
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return(milliseconds);
}*/

void getProperties(){
    int nDevices;
  hipGetDeviceCount(&nDevices);
  
  printf("Number of devices: %d\n", nDevices);
  
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
    printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
    printf("  minor-major: %d-%d\n", prop.minor, prop.major);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
    printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
  }
}
